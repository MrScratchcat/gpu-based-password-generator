
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>

const int NUM_PASSWORDS = 1024;
__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!@#$%^&*()-_=+[]{}|;:,.<>?";

__global__ void generatePasswords(char *passwords, int charsetLength, int minLen, int maxLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);

    int passLen = minLen + hiprand(&state) % (maxLen - minLen + 1);

    for (int i = 0; i < passLen; ++i) {
        passwords[idx * (maxLen + 1) + i] = charset[hiprand(&state) % charsetLength];
    }
    passwords[idx * (maxLen + 1) + passLen] = '\0'; // Null-terminate the string
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <min_length> <max_length>\n", argv[0]);
        return 1;
    }

    int minLen = atoi(argv[1]);
    int maxLen = atoi(argv[2]);

    if (minLen > maxLen || minLen < 1 or maxLen < 1) {
        printf("Invalid length parameters.\n");
        return 1;
    }

    char *d_passwords;
    char *h_passwords = (char *)malloc(NUM_PASSWORDS * (maxLen + 1) * sizeof(char));

    hipMalloc(&d_passwords, NUM_PASSWORDS * (maxLen + 1) * sizeof(char));

    while (true) {  // Repeat the generation process indefinitely
        generatePasswords<<<1, NUM_PASSWORDS>>>(d_passwords, sizeof(charset) - 1, minLen, maxLen);
        hipDeviceSynchronize();

        hipMemcpy(h_passwords, d_passwords, NUM_PASSWORDS * (maxLen + 1) * sizeof(char), hipMemcpyDeviceToHost);

        for (int i = 0; i < NUM_PASSWORDS; ++i) {
            printf("%s\n", h_passwords + i * (maxLen + 1));
        }

        // Optional: Add a delay if needed
        // sleep(1);
    }

    // Cleanup (unreachable in this version of the code)
    hipFree(d_passwords);
    free(h_passwords);

    return 0;
}
