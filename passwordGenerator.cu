
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>

const int NUM_PASSWORDS = 1024;
const int blockSize = 1024; // Max threads per block (depends on your GPU's capability)
const int gridSize = 1000 * (NUM_PASSWORDS + blockSize - 1) / blockSize; // 10x larger grid size
__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!@#$%^&*()-_=+[]{}|;:,.<>?";

__global__ void generatePasswords(char *passwords, int charsetLength, int minLen, int maxLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NUM_PASSWORDS * gridSize) return; // Guard to avoid out-of-bounds work

    hiprandState state;
    hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);

    int passLen = minLen + hiprand(&state) % (maxLen - minLen + 1);

    for (int i = 0; i < passLen; ++i) {
        passwords[idx * (maxLen + 1) + i] = charset[hiprand(&state) % charsetLength];
    }
    passwords[idx * (maxLen + 1) + passLen] = '\0'; // Null-terminate the string
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <min_length> <max_length>\n", argv[0]);
        return 1;
    }

    int minLen = atoi(argv[1]);
    int maxLen = atoi(argv[2]);

    if (minLen > maxLen || minLen < 1 || maxLen < 1) {
        printf("Invalid length parameters.\n");
        return 1;
    }

    char *d_passwords;
    char *h_passwords = (char *)malloc(NUM_PASSWORDS * gridSize * (maxLen + 1) * sizeof(char));

    hipMalloc(&d_passwords, NUM_PASSWORDS * gridSize * (maxLen + 1) * sizeof(char));

    while (true) {  // Repeat the generation process indefinitely
        generatePasswords<<<gridSize, blockSize>>>(d_passwords, sizeof(charset) - 1, minLen, maxLen);
        hipDeviceSynchronize();

        hipMemcpy(h_passwords, d_passwords, NUM_PASSWORDS * gridSize * (maxLen + 1) * sizeof(char), hipMemcpyDeviceToHost);

        for (int i = 0; i < NUM_PASSWORDS * gridSize; ++i) {
            printf("%s\n", h_passwords + i * (maxLen + 1));
        }

        // Optional: Add a delay if needed
        // sleep(1);
    }

    // Cleanup (unreachable in this version of the code)
    hipFree(d_passwords);
    free(h_passwords);

    return 0;
}
